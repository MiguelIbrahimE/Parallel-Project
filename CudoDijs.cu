#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <hip/hip_runtime.h>

#define V 20000

__global__ void dijkstra_kernel(int* graph, int* dist, int* sptSet, int src)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid == src) {
        dist[tid] = 0;
        sptSet[tid] = 1;
    }
    else {
        dist[tid] = INT_MAX;
        sptSet[tid] = 0;
    }

    __syncthreads();

    for (int count = 0; count < V - 1; count++) {
        int u = -1;
        int minDist = INT_MAX;

        for (int v = 0; v < V; v++) {
            if (sptSet[v] == 0 && dist[v] < minDist) {
                u = v;
                minDist = dist[v];
            }
        }

        sptSet[u] = 1;

        for (int v = 0; v < V; v++) {
            int weight = graph[u * V + v];
            if (weight && !sptSet[v] && dist[u] != INT_MAX && dist[u] + weight < dist[v]) {
                dist[v] = dist[u] + weight;
            }
        }

        __syncthreads();
    }
}

void printSolution(int dist[])
{
    printf("Vertex \t Distance from Source\n");
    for (int i = 0; i < V; i++)
        printf("%d \t %d\n", i, dist[i]);
}

int main()
{
    int* graph_h = (int*)malloc(V * V * sizeof(int));
    int* dist_h = (int*)malloc(V * sizeof(int));
    int* sptSet_h = (int*)malloc(V * sizeof(int));

    // Generate random graph with 1000 vertices and 10000 edges
    for (int i = 0; i < V; i++) {
        for (int j = 0; j < V; j++) {
            if (i == j)
                graph_h[i * V + j] = 0;
            else if (rand() % 10 == 0)
                graph_h[i * V + j] = rand() % 1000;
            else
                graph_h[i * V + j] = 0;
        }
    }

    int* graph_d, *dist_d, *sptSet_d;
    hipMalloc(&graph_d, V * V * sizeof(int));
    hipMalloc(&dist_d, V * sizeof(int));
    hipMalloc(&sptSet_d, V * sizeof(int));

    hipMemcpy(graph_d, graph_h, V * V * sizeof(int), hipMemcpyHostToDevice);

    // Measure the start time
    clock_t start_time = clock();

    dijkstra_kernel<<<1, V>>>(graph_d, dist_d, sptSet_d, 0);

    hipMemcpy(dist_h, dist_d, V * sizeof(int), hipMemcpyDeviceToHost);

    // Measure the end time
    clock_t end_time = clock();

    // Calculate the time taken
    double time_taken = (double)(end_time - start_time) / CLOCKS_PER_SEC;

    printSolution(dist_h);
    printf("Dijkstra's algorithm took %f seconds to execute.\n", time_taken);

    hipFree(graph_d);
    hipFree(dist_d);
    hipFree(sptSet_d);

    free(graph_h);
    free(dist_h);
    free(sptSet_h);
    return 0;
    }
